#include "memoryAllocator.h"
#include <iostream>
#include <cstring>

// Helper function to check CUDA errors
inline void checkCudaError(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n" << "CUDA error message = " << hipGetErrorString(result) << std::endl;
        hipDeviceReset();
        exit(99);
    }
}

#define CUDA_CHECK(val) checkCudaError((val), #val, __FILE__, __LINE__)

void* allocate_memory(size_t size) {
    void* devicePtr;
    hipError_t status = hipMalloc(&devicePtr, size);
    if (status != hipSuccess) {
        CUDA_CHECK(status); // This will handle the error and exit if there is an issue
    } else {
        std::cout << "Allocated " << size << " bytes on GPU at address " << devicePtr << std::endl;
    }
    return device_ptr;
}

void deallocate_memory(void* ptr) {
    hipError_t status = hipFree(ptr);
    if (status != hipSuccess) {
        CUDA_CHECK(status); // This will handle the error and exit if there is an issue
    } else {
        std::cout << "Deallocated memory at address " << ptr << std::endl;
    }
}
