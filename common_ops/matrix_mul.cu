#include "hip/hip_runtime.h"
#include "matrix_mul.h"
#include <iostream>

__global__ void matrixMulKernel(const int *a, const int *b, int *c, int N) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < N && col < N) {
    int tmp = 0;
    for (int k = 0; k < N; ++k) {
      tmp += a[row * N + k] * b[k * N + col];
    }
    c[row * N + col] = tmp;
  }
}

void matrix_mul(const std::vector<int>& a, const std::vector<int>& b, std::vector<int>& c, int N) {
  size_t bytes = N * N * sizeof(int);
  int *d_a, *d_b, *d_c;

  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

  matrixMulKernel<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, N);

  hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
