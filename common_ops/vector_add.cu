#include "hip/hip_runtime.h"
// vector_add.cu
#include "vector_add.h"
#include <iostream>

__global__ void vectorAdd(int *a, int *b, int *c, int N) {
  // Calculate global thread thread ID
  int tid = GET_GLOBAL_THREAD_IDX();

  // Boundary check
  if (tid < N) {
    c[tid] = a[tid] + b[tid];
  }
}

void vector_add(const int* a, const int* b, int* result, int size) {
    size_t bytes = size * sizeof(int);
    int *d_a, *d_b, *d_result;

    // Allocate memory on the GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_result, bytes);

    // Copy input vectors from host memory to GPU buffers
    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice);

    // Determine grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Launch vector addition kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_result, size);

    // Copy result from GPU buffer to host memory
    hipMemcpy(result, d_result, bytes, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
}
