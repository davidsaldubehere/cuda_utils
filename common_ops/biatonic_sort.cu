#include "hip/hip_runtime.h"
#include "biatonic_sort.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <algorithm>

#define MAX_THREADS_PER_BLOCK 1024

//GPU Kernel Implementation of Bitonic Sort
__global__ void bitonicSortGPU(int* arr, int j, int k)
{
    unsigned int i, ij;
    i = GET_GLOBAL_THREAD_IDX();
    ij = i ^ j;
    if (ij > i)
    {    if ((i & k) == 0)
        {    if (arr[i] > arr[ij])
            {   int temp = arr[i];
                arr[i] = arr[ij];
                arr[ij] = temp;
            }    }
        else
        {    if (arr[i] < arr[ij])
            {   int temp = arr[i];
                arr[i] = arr[ij];
                arr[ij] = temp;
            }    }    }    }



//MAIN PROGRAM
void sort(int* arr, int size) {       
    int* gpuArrmerge;
    int* gpuArrbiton;
    int* gpuTemp;
    // Allocate memory on GPU
    hipMalloc((void**)&gpuArrmerge, size * sizeof(int));
    hipMalloc((void**)&gpuTemp, size * sizeof(int));
    hipMalloc((void**)&gpuArrbiton, size * sizeof(int));
    // Copy the input array to GPU memory
    hipMemcpy(gpuArrmerge, arr, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpuArrbiton, arr, size * sizeof(int), hipMemcpyHostToDevice);
    //Set number of threads and blocks for kernel calls
    int threadsPerBlock = MAX_THREADS_PER_BLOCK;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    int j, k;
    //Time the run and call GPU Bitonic Kernel
    for (k = 2; k <= size; k <<= 1){
        for (j = k >> 1; j > 0; j = j >> 1){
            bitonicSortGPU << <blocksPerGrid, threadsPerBlock >> > (gpuArrbiton, j, k);
        }    }
    hipMemcpy(arr, gpuArrbiton, size * sizeof(int), hipMemcpyDeviceToHost);
    //Destroy all variables
    delete[] arr;
    //End
    hipFree(gpuArrmerge);
    hipFree(gpuArrbiton);
    hipFree(gpuTemp);
}
