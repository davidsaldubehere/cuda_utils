#include "hip/hip_runtime.h"
#include "biatonic_sort.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <algorithm>

#define MAX_THREADS_PER_BLOCK 1024

//GPU Kernel Implementation of Bitonic Sort
__global__ void bitonicSortGPU(int* arr, int j, int k)
{
    unsigned int i, ij;

    i = GET_GLOBAL_THREAD_IDX();

    ij = i ^ j;

    if (ij > i)
    {
        if ((i & k) == 0)
        {
            if (arr[i] > arr[ij])
            {
                int temp = arr[i];
                arr[i] = arr[ij];
                arr[ij] = temp;
            }
        }
        else
        {
            if (arr[i] < arr[ij])
            {
                int temp = arr[i];
                arr[i] = arr[ij];
                arr[ij] = temp;
            }
        }
    }
}


//Function to print array
void printArray(int* arr, int size) 
{
    for (int i = 0; i < size; ++i)
        std::cout << arr[i] << " ";
    std::cout << std::endl;
}

//Automated function to check if array is sorted
bool isSorted(int* arr, int size) 
{
    for (int i = 1; i < size; ++i) 
    {
        if (arr[i] < arr[i - 1])
            return false;
    }
    return true;
}

//MAIN PROGRAM
int main()
{   
    int size = 128;
    
    //Create CPU based Arrays
    int* arr = new int[size];
    int* carr = new int[size];
    int* temp = new int[size];

    //Create GPU based arrays
    int* gpuArrmerge;
    int* gpuArrbiton;
    int* gpuTemp;

    // Initialize the array with random values
    srand(static_cast<unsigned int>(time(nullptr)));
    for (int i = 0; i < size; ++i) 
    {
        arr[i] = rand() % 100;
        carr[i] = arr[i];
    }

    //Print unsorted array 
    std::cout << "\n\nUnsorted array: ";
    if (size <= 100) 
    {
        printArray(arr, size);
    }
    else 
    {
        printf("\nToo Big to print. Check Variable. Automated isSorted Checker will be implemented\n");
    }

    // Allocate memory on GPU
    hipMalloc((void**)&gpuArrmerge, size * sizeof(int));
    hipMalloc((void**)&gpuTemp, size * sizeof(int));
    hipMalloc((void**)&gpuArrbiton, size * sizeof(int));

    // Copy the input array to GPU memory
    hipMemcpy(gpuArrmerge, arr, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpuArrbiton, arr, size * sizeof(int), hipMemcpyHostToDevice);

    //Set number of threads and blocks for kernel calls
    int threadsPerBlock = MAX_THREADS_PER_BLOCK;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;


    int j, k;
    //Time the run and call GPU Bitonic Kernel
    hipEventRecord(startGPU);
    for (k = 2; k <= size; k <<= 1)
    {
        for (j = k >> 1; j > 0; j = j >> 1)
        {
            bitonicSortGPU << <blocksPerGrid, threadsPerBlock >> > (gpuArrbiton, j, k);
        }
    }
    //Transfer Sorted array back to CPU
    hipMemcpy(arr, gpuArrbiton, size * sizeof(int), hipMemcpyDeviceToHost);

    // Display sorted GPU array
    std::cout << "\n\nSorted GPU array: ";
    if (size <= 100) 
    {
        printArray(arr, size);
    }
    else {
        printf("\nToo Big to print. Check Variable. Automated isSorted Checker will be implemented\n");
    }

    
    //Run the array with the automated isSorted checker
    if (isSorted(arr, size))
        std::cout << "\n\nSORT CHECKER RUNNING - SUCCESFULLY SORTED GPU ARRAY" << std::endl;
    else
        std::cout << "SORT CHECKER RUNNING - !!! FAIL !!!" << std::endl;

    //Destroy all variables
    delete[] arr;
    delete[] carr;
    delete[] temp;

    //End
    hipFree(gpuArrmerge);
    hipFree(gpuArrbiton);
    hipFree(gpuTemp);

    std::cout << "\n------------------------------------------------------------------------------------\n||||| END. YOU MAY RUN THIS AGAIN |||||\n------------------------------------------------------------------------------------";
    return 0;
}